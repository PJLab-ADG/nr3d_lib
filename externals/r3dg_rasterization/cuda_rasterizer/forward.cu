#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for
	// "Differentiable Point-Based Radiance Fields for
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002).
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}


// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
    // Create scaling matrix
    glm::mat3 S = glm::mat3(1.0f);
    S[0][0] = mod * scale.x;
    S[1][1] = mod * scale.y;
    S[2][2] = mod * scale.z;

    // Normalize quaternion to get valid rotation
    glm::vec4 q = rot;// / glm::length(rot);
    float r = q.x;
    float x = q.y;
    float y = q.z;
    float z = q.w;

    // Compute rotation matrix from quaternion
    glm::mat3 R = glm::mat3(
            1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
            2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
            2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
    );

    glm::mat3 M = S * R;

    // Compute 3D world covariance matrix Sigma
    glm::mat3 Sigma = glm::transpose(M) * M;

    // Covariance is symmetric, only store upper right
    cov3D[0] = Sigma[0][0];
    cov3D[1] = Sigma[0][1];
    cov3D[2] = Sigma[0][2];
    cov3D[3] = Sigma[1][1];
    cov3D[4] = Sigma[1][2];
    cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	int* radii,
	float2* points_xy_image,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters.
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// Invert covariance (EWA algorithm)
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	float det_inv = 1.f / det;
	float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles.
	float mid = 0.5f * (cov.x + cov.z);
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	uint2 rect_min, rect_max;
	getRect(point_image, my_radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
	radii[idx] = my_radius;

	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one float4
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] };
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	const int S, int W, int H,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ depths,
	const float* __restrict__ features,
	const float* __restrict__ colors,
	const float4* __restrict__ conic_opacity,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color,
	float* __restrict__ out_opacity,
	float* __restrict__ out_depth,
	float* __restrict__ out_feature)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 }, F[33] = { 0 }, Depth = 0, Opacity = 0;

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix).
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

            float weight = alpha * T;
			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += colors[collected_id[j] * CHANNELS + ch] * weight;

			for (int ch = 0; ch < S; ch++)
				F[ch] += features[collected_id[j] * S + ch] * weight;

			Depth += depths[collected_id[j]] * weight;
            Opacity += weight;

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		for (int ch = 0; ch < S; ch++)
			out_feature[ch * H * W + pix_id] = F[ch];
		out_depth[pix_id] = Depth;
		out_opacity[pix_id] = Opacity;
	}
}


__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderSurfaceXYZCUDA(
    const int W, const int H,
    const float* viewmatrix,
    const float focal_x, const float focal_y,
    const float cx, const float cy,
    const float tan_fovx, const float tan_fovy,
    const float* __restrict__ opacities,
    const float* __restrict__ depths,
    float* __restrict__ normals,
    float* __restrict__ surface_xyz)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
    if (!inside) return;
	uint32_t pix_id = W * pix.y + pix.x;
    uint32_t HW = H * W;
	float depth = depths[pix_id] / fmaxf(opacities[pix_id], 0.0000001f);
// 	float depth = depths[pix_id];
    surface_xyz[pix_id] =  (pix.x - cx) / focal_x * depth;
    surface_xyz[HW + pix_id] = (pix.y - cy) / focal_y * depth;
    surface_xyz[2 * HW + pix_id] = depth;
}

__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderPseudoNormalCUDA(
    const int W, const int H,
    const float* viewmatrix,
    const float focal_x, const float focal_y,
    const float cx, const float cy,
    const float tan_fovx, const float tan_fovy,
    const float* __restrict__ depths,
    float* __restrict__ normals,
    const float* __restrict__ surface_xyz)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
    if (!inside) return;
    uint32_t HW = H * W;
    float gradient_a[3], gradient_b[3];
	uint32_t pix_id00 = W * (pix.y==0?0:pix.y-1) + (pix.x==0?0:pix.x-1);
	uint32_t pix_id01 = W * (pix.y==0?0:pix.y-1) + pix.x;
	uint32_t pix_id02 = W * (pix.y==0?0:pix.y-1) + (pix.x==W-1?W-1:pix.x+1);
	uint32_t pix_id10 = W * pix.y + (pix.x==0?0:pix.x-1);
	uint32_t pix_id11 = W * pix.y + pix.x;
	uint32_t pix_id12 = W * pix.y + (pix.x==W-1?W-1:pix.x+1);
	uint32_t pix_id20 = W * (pix.y==H-1?H-1:pix.y+1) + (pix.x==0?0:pix.x-1);
	uint32_t pix_id21 = W * (pix.y==H-1?H-1:pix.y+1) + pix.x;
	uint32_t pix_id22 = W * (pix.y==H-1?H-1:pix.y+1) + (pix.x==W-1?W-1:pix.x+1);

    float xyz00[3] = {surface_xyz[pix_id00],surface_xyz[HW + pix_id00],surface_xyz[2 * HW + pix_id00]};
    float xyz01[3] = {surface_xyz[pix_id01],surface_xyz[HW + pix_id01],surface_xyz[2 * HW + pix_id01]};
    float xyz02[3] = {surface_xyz[pix_id02],surface_xyz[HW + pix_id02],surface_xyz[2 * HW + pix_id02]};
    float xyz10[3] = {surface_xyz[pix_id10],surface_xyz[HW + pix_id10],surface_xyz[2 * HW + pix_id10]};
    //float xyz11[3] = {surface_xyz[pix_id11],surface_xyz[HW + pix_id11],surface_xyz[2 * HW + pix_id11]};
    float xyz12[3] = {surface_xyz[pix_id12],surface_xyz[HW + pix_id12],surface_xyz[2 * HW + pix_id12]};
    float xyz20[3] = {surface_xyz[pix_id20],surface_xyz[HW + pix_id20],surface_xyz[2 * HW + pix_id20]};
    float xyz21[3] = {surface_xyz[pix_id21],surface_xyz[HW + pix_id21],surface_xyz[2 * HW + pix_id21]};
    float xyz22[3] = {surface_xyz[pix_id22],surface_xyz[HW + pix_id22],surface_xyz[2 * HW + pix_id22]};
    for (int i=0;i<3;i++){
        gradient_a[i] = -0.125f * xyz00[i] + 0.125f * xyz02[i] - 0.25f * xyz10[i] + 0.25f * xyz12[i] - 0.125f * xyz20[i] + 0.125f * xyz22[i];
    }

    for (int i=0;i<3;i++){
        gradient_b[i] = -0.125f * xyz00[i] - 0.25f * xyz01[i] - 0.125f * xyz02[i] + 0.125f * xyz20[i] + 0.25f * xyz21[i] + 0.125f * xyz22[i];
    }
    float normal[3] = {
        gradient_a[1] * gradient_b[2] - gradient_a[2] * gradient_b[1],
        -gradient_a[0] * gradient_b[2] + gradient_a[2] * gradient_b[0],
        gradient_a[0] * gradient_b[1] - gradient_a[1] * gradient_b[0]
    };
    float norm = sqrtf(normal[0]*normal[0]+normal[1]*normal[1]+normal[2]*normal[2]);
    if (norm <= 0.00000f){
        return;
    }

    normal[0] = -normal[0] / norm;
    normal[1] = -normal[1] / norm;
    normal[2] = -normal[2] / norm;

    normals[pix_id11] = viewmatrix[0] * normal[0] + viewmatrix[1] * normal[1] + viewmatrix[2] * normal[2];
    normals[HW + pix_id11] = viewmatrix[4] * normal[0] + viewmatrix[5] * normal[1] + viewmatrix[6] * normal[2];
    normals[2 * HW + pix_id11] = viewmatrix[8] * normal[0] + viewmatrix[9] * normal[1] + viewmatrix[10] * normal[2];
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	const int S, int W, int H,
	const float2* means2D,
	const float* depths,
	const float* features,
	const float* colors,
	const float4* conic_opacity,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color,
	float* out_opacity,
	float* out_depth,
	float* out_feature
	)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> > (
		ranges,
		point_list,
		S, W, H,
		means2D,
		depths,
		features,
		colors,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color,
		out_opacity,
		out_depth,
		out_feature);
}

void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	float2* means2D,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	preprocessCUDA<NUM_CHANNELS> <<<(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix,
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		cov3Ds,
		rgb,
		conic_opacity,
		grid,
		tiles_touched,
		prefiltered
		);
}

void FORWARD::render_xyz(
	const dim3 grid, dim3 block,
	const int W, const int H,
    const float* viewmatrix,
    const float focal_x, const float focal_y,
    const float cx, const float cy,
    const float tan_fovx, const float tan_fovy,
    const float* opacities,
    const float* depths,
    float* normals,
    float* surface_xyz)
{
	renderSurfaceXYZCUDA << <grid, block >> > (
		W, H,
        viewmatrix,
        focal_x, focal_y,
        cx, cy,
        tan_fovx, tan_fovy,
        opacities,
        depths,
        normals,
        surface_xyz);
}

void FORWARD::render_pseudo_normal(
	const dim3 grid, dim3 block,
	const int W, const int H,
    const float* viewmatrix,
    const float focal_x, const float focal_y,
    const float cx, const float cy,
    const float tan_fovx, const float tan_fovy,
    const float* opacities,
    const float* depths,
    float* normals,
    float* surface_xyz)
{
    renderPseudoNormalCUDA << <grid, block >> > (
		W, H,
        viewmatrix,
        focal_x, focal_y,
        cx, cy,
        tan_fovx, tan_fovy,
        depths,
        normals,
        surface_xyz);
}